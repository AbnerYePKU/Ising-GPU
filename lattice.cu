#include "hip/hip_runtime.h"
#include "lattice.h"

using namespace std;

lattice::lattice(const int N_): N(N_)
{
    hipMalloc(&a, N*N*sizeof(int));
    hipMalloc(&obs, 2*sizeof(int));
    hipMalloc(&real_dist, N*N*sizeof(double));
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937); // initialize rng
    hiprandSetPseudoRandomGeneratorSeed(gen, 0); // set seed
}

lattice::~lattice()
{
    hipFree(a);
    hipFree(obs);
    hipFree(real_dist);
}

__global__ void _initialize_1(int *a)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    a[i] = 1;
}

__global__ void _initialize_r(int *a, double *real_dist)
{
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    a[i] = (real_dist[i]>0.5)?(1):(-1);
}

void lattice::initialize(char option)
{
    switch(option)
    {
        case '1': // all +1 spins
            _initialize_1<<<N,N>>>(a);
            return;
        case 'r': // random +1/-1 spins
            hiprandGenerateUniformDouble(gen, real_dist, N*N);
            _initialize_r<<<N,N>>>(a, real_dist);
            return;
    }
}

__global__ void _S(const int *a, int *obs)
{
    // ---- indexes ---- //
    const int N = blockDim.x;
    int i = blockIdx.x;
    int i_ = _res(i+1,N)*N;
    i *= N;
    int j = threadIdx.x;
    int j_ = _res(j+1,N);
    // ---- spins to use ---- //
    int spin_1 = a[i+j];
    int spin_2 = a[i_+j]+a[i+j_];
    // ---- calculate observables ---- //
    atomicAdd(obs+0, spin_1);
    atomicAdd(obs+1, spin_1*spin_2);
}

void S(const lattice &sigma, int *obs)
{
    const int N = sigma.N;
    hipMemset(sigma.obs, 0, 2*sizeof(int));
    _S<<<N,N>>>(sigma.a, sigma.obs);
    hipMemcpy(obs, sigma.obs, 2*sizeof(int), hipMemcpyDeviceToHost);
}

std::ostream & operator<<(std::ostream &os, const lattice &sigma)
{
    const int N = sigma.N;
    int i,j, a[N*N];
    // memory copy from device to host
    hipMemcpy(a, sigma.a, N*N*sizeof(int), hipMemcpyDeviceToHost);
    for(i=0;i<N*N;i+=N){
        for(j=0;j<N;j++){
            os << a[i+j] << " ";
        }
        os << endl;
    }
    return os;
}
